#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"

#include "SimpleOptimizations.cuh"

__global__ void
transformVectorByMatrix4( const  float4 *pInput, const int *pIndex, float4 *pMatrix, float4 *pOutput,  int sizeMax,  const float *pWeight)
{
	const int indexBase = ( gridDim.x * blockIdx.y + blockIdx.x ) * blockDim.x + threadIdx.x;

	if( indexBase >= sizeMax )
		return;

	int index=indexBase;
#if SIZE_BLOCK_STATIC
		for( ; index<sizeMax; index+=blockDim.x * gridDim.x )
#endif
		{
			int offset = pIndex[index*SIZE_PER_BONE+0]*4 ;
			float weight = pWeight[index*SIZE_PER_BONE+0] ;
			float4 weight4 = make_float4( weight,weight,weight,weight ) ;

			float4 m0 = pMatrix[offset+0] * weight4 ;
			float4 m1 = pMatrix[offset+1] * weight4 ;
			float4 m2 = pMatrix[offset+2] * weight4 ;
			float4 m3 = pMatrix[offset+3] * weight4 ;

			for(int i=1;i<SIZE_PER_BONE; i++)
			{
				offset = pIndex[index*SIZE_PER_BONE+i]*4 ;
				weight = pWeight[index*SIZE_PER_BONE+i] ;
				weight4 = make_float4( weight, weight, weight, weight ) ;

				m0 += pMatrix[offset+0] * weight4 ;
				m1 += pMatrix[offset+1] * weight4 ;
				m2 += pMatrix[offset+2] * weight4 ;
				m3 += pMatrix[offset+3] * weight4 ;
			}

			float4 pIn = pInput[index];
			float4 px = make_float4(pIn.x, pIn.x, pIn.x, pIn.x) ;
			float4 py = make_float4(pIn.y, pIn.y, pIn.y, pIn.y) ;
			float4 pz = make_float4(pIn.z, pIn.z, pIn.z, pIn.z) ;

			pOutput[index] = px * m0 + py * m1 + pz * m2 + m3;
		}
}

__global__ void
transformVectorByMatrix4( const  Vector4 *pInput, const Vector1i *pIndex, Vector4 *pMatrix, Vector4 *pOutput,  int sizeMax,  const Vector1 *pWeight)
{
	const int indexBase = ( gridDim.x * blockIdx.y + blockIdx.x ) * blockDim.x + threadIdx.x;

	if( indexBase >= sizeMax )
		return;

	int index=indexBase;
#if SIZE_BLOCK_STATIC
		for( ; index<sizeMax; index+=blockDim.x * gridDim.x )
#endif
		{
			int offset = pIndex[index*SIZE_PER_BONE+0].x*4 ;
			Vector1 weight = pWeight[index*SIZE_PER_BONE+0] ;
			Vector4 weight4 = make_vector4( weight.x,weight.x,weight.x,weight.x ) ;

			Vector4 m0 = pMatrix[offset+0] * weight4 ;
			Vector4 m1 = pMatrix[offset+1] * weight4 ;
			Vector4 m2 = pMatrix[offset+2] * weight4 ;
			Vector4 m3 = pMatrix[offset+3] * weight4 ;

			for(int i=1;i<SIZE_PER_BONE; i++)
			{
				offset = pIndex[index*SIZE_PER_BONE+i].x*4 ;
				weight = pWeight[index*SIZE_PER_BONE+i] ;
				weight4 = make_vector4( weight.x,weight.x,weight.x,weight.x ) ;

				m0 += pMatrix[offset+0] * weight4 ;
				m1 += pMatrix[offset+1] * weight4 ;
				m2 += pMatrix[offset+2] * weight4 ;
				m3 += pMatrix[offset+3] * weight4 ;
			}

			Vector4 pIn = pInput[index];
			Vector4 px = make_vector4(pIn.x, pIn.x, pIn.x, pIn.x) ;
			Vector4 py = make_vector4(pIn.y, pIn.y, pIn.y, pIn.y) ;
			Vector4 pz = make_vector4(pIn.z, pIn.z, pIn.z, pIn.z) ;

			pOutput[index] = px * m0 + py * m1 + pz * m2 + m3;
		}
}

__global__ void
transformVectorByMatrix4One( const float4 *pInput, const int1 *pIndex, float4 *pMatrix, float4 *pOutput,  int sizeMax,  const float1 *pWeight)
{
	//size_t index = get_global_id(0) + get_global_id(1) *get_global_size(0);
	const int indexBase = ( gridDim.x * blockIdx.y + blockIdx.x ) * blockDim.x + threadIdx.x;

	if( indexBase >= sizeMax )
		return;

	int index=indexBase;
#if SIZE_BLOCK_STATIC
		for( ; index<sizeMax; index+=blockDim.x * gridDim.x )
#endif
		{

			float4 pIn = pInput[index];
			float4 px = make_float4(pIn.x, pIn.x, pIn.x, pIn.x) ;
			float4 py = make_float4(pIn.y, pIn.y, pIn.y, pIn.y) ;
			float4 pz = make_float4(pIn.z, pIn.z, pIn.z, pIn.z) ;

			int offset = pIndex[index].x*4 ;

			float4 m0 = pMatrix[offset+0] ;
			float4 m1 = pMatrix[offset+1] ;
			float4 m2 = pMatrix[offset+2] ;
			float4 m3 = pMatrix[offset+3] ;

			pOutput[index] = px * m0 + py * m1 + pz * m2 + m3 ;
		}
}

__global__ void
transformVectorByMatrix4One( const Vector4 *pInput, const Vector1i *pIndex, Vector4 *pMatrix, Vector4 *pOutput,  int sizeMax,  const Vector1 *pWeight)
{
	//size_t index = get_global_id(0) + get_global_id(1) *get_global_size(0);
	const int indexBase = ( gridDim.x * blockIdx.y + blockIdx.x ) * blockDim.x + threadIdx.x;

	if( indexBase >= sizeMax )
		return;

	int index=indexBase;
#if SIZE_BLOCK_STATIC
		for( ; index<sizeMax; index+=blockDim.x * gridDim.x )
#endif
		{

			Vector4 pIn = pInput[index];
			Vector4 px = make_vector4(pIn.x, pIn.x, pIn.x, pIn.x) ;
			Vector4 py = make_vector4(pIn.y, pIn.y, pIn.y, pIn.y) ;
			Vector4 pz = make_vector4(pIn.z, pIn.z, pIn.z, pIn.z) ;

			int offset = pIndex[index].x*4 ;

			Vector4 m0 = pMatrix[offset+0] ;
			Vector4 m1 = pMatrix[offset+1] ;
			Vector4 m2 = pMatrix[offset+2] ;
			Vector4 m3 = pMatrix[offset+3] ;

			pOutput[index] = px * m0 + py * m1 + pz * m2 + m3 ;
		}
}

/* �������任
pVertex  : ����
pMatrix : ����
*/
template<typename F4>
__device__ void transformVec3ByMatrix4(F4* pVertexIn, float1 pMatrix[], F4* pVertexOut)
{
	F4 vertexIn = *pVertexIn;
	F4 vertexOut;
	vertexOut.x = vertexIn.x * pMatrix[0].x + vertexIn.y * pMatrix[1].x + vertexIn.z * pMatrix[2].x + pMatrix[3].x ; 
	vertexOut.y = vertexIn.x * pMatrix[1*4+0].x + vertexIn.y * pMatrix[1*4+1].x + vertexIn.z * pMatrix[1*4+2].x + pMatrix[1*4+3].x  ; 
	vertexOut.z = vertexIn.x * pMatrix[2*4+0].x + vertexIn.y * pMatrix[2*4+1].x + vertexIn.z * pMatrix[2*4+2].x + pMatrix[2*4+3].x  ;
	*pVertexOut = vertexOut;
}
template<typename F4>
__device__ void transformVec3ByMatrix4(F4* pVertexIn, F4 pMatrix[], F4* pVertexOut)
{
	F4 vertexIn = *pVertexIn;
	F4 vertexOut;
	vertexOut.x = vertexIn.x * pMatrix[0].x + vertexIn.y * pMatrix[0].y + vertexIn.z * pMatrix[0].z + pMatrix[0].w ; 
	vertexOut.y = vertexIn.x * pMatrix[1].x + vertexIn.y * pMatrix[1].y + vertexIn.z * pMatrix[1].z + pMatrix[1].w  ; 
	vertexOut.z = vertexIn.x * pMatrix[2].x + vertexIn.y * pMatrix[2].y + vertexIn.z * pMatrix[2].z + pMatrix[2].w  ;
	*pVertexOut = vertexOut;
}

template<typename F4>
__device__ void transformVec3ByMatrix4_f4(F4* pVertexIn, float4 pMatrix[], F4* pVertexOut)
{
	F4 vertexIn = *pVertexIn;
	F4 vertexOut;
	vertexOut.x = vertexIn.x * pMatrix[0].x + vertexIn.y * pMatrix[0].y + vertexIn.z * pMatrix[0].z + pMatrix[0].w ; 
	vertexOut.y = vertexIn.x * pMatrix[1].x + vertexIn.y * pMatrix[1].y + vertexIn.z * pMatrix[1].z + pMatrix[1].w  ; 
	vertexOut.z = vertexIn.x * pMatrix[2].x + vertexIn.y * pMatrix[2].y + vertexIn.z * pMatrix[2].z + pMatrix[2].w  ;
	*pVertexOut = vertexOut;
}

	// ����������
template<typename F4>
__device__ void indexByFloat44( F4* pBuffer , F4* pMat , int index )
	{
		for(int j=0; j<MATRIX_SIZE_LINE; j++){
			pMat[j] = pBuffer[index * MATRIX_SIZE_LINE + j];
		}
	}


__global__ void updateVectorByMatrix(float4* pVertexIn, int size, float1* pMatrix, float4* pVertexOut)
{
	const int indexBase = ( gridDim.x * blockIdx.y + blockIdx.x ) * blockDim.x + threadIdx.x;

		for( int i=indexBase; i<size; i+=blockDim.x * gridDim.x ){

		float4   matrix[MATRIX_SIZE_LINE];

		// ��ȡ����������ʼ�Ķ�������
		float4   vertexIn = pVertexIn[i];

		// ��ȡ�������������Ӧ�ľ���
		int      matrixIndex = int(vertexIn.w + 0.5);// float to int

		
		indexByFloat44( (float4*)pMatrix, matrix, matrixIndex );

		// ִ�в�����������ִ�о���任���õ�������
		transformVec3ByMatrix4( &vertexIn, matrix, pVertexOut+i);
	}//for
}

extern "C" bool
runCUDADevice( const float *pInput, const int *pIndex, float *pMatrix, float *pOutput,  int sizeMax,  const float *pWeight )
{
	int nCountThreadsPerBlock = SIZE_THREAD_X;
    dim3 block( nCountThreadsPerBlock, 1, 1);

#if SIZE_BLOCK_STATIC
	dim3 grid( SIZE_BLOCK_X, 1, 1);
#else
	int nCountBlocks = (sizeMax + nCountThreadsPerBlock - 1) / nCountThreadsPerBlock ;
	dim3 grid( nCountBlocks, 1, 1);
#endif

    // execute the kernel
#if SIZE_PER_BONE==1
    transformVectorByMatrix4One<<< grid, block >>>( (FLOAT4*)pInput, (INT1*)pIndex, (FLOAT4*)pMatrix, (FLOAT4*)pOutput, sizeMax, (FLOAT1*)pWeight );
#else
    transformVectorByMatrix4<<< grid, block >>>( (FLOAT4*)pInput, (int*)pIndex, (FLOAT4*)pMatrix, (FLOAT4*)pOutput, sizeMax, (float*)pWeight );
#endif
	//updateVectorByMatrix<<< grid, block >>>( (float4*)pInput, sizeMax, (float1*)pMatrix, (float4*)pOutput );

	return true;
}